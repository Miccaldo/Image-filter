#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "CImg-2.5.0/CImg.h"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <cmath>
#include <time.h>
#include <windows.h> 
#include <stddef.h>

#define cimg_use_jpeg 1

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define WRITE_FLATTEN_MATRIX	false
#define GET_FLATTEN_MATRIX		true

#include ""
using namespace cimg_library;
using namespace std;


void filter(double *matrix, double *copy, double *mask, int width, int height, int channels, int maskLength, double normalize);
void writeOrGetMatrix(CImg<double> &image, double *matrix, bool type);
double getNormalize(double *mask, int size);
void blow(double *mask, int size);
void sharpen(double *mask);
void prominence(double *mask);
void gradient(double *mask);
int init();
int getMaskLength(int filterType);
void setFilter(double *mask, int filterType);
void laplace(double *mask);


__global__ void dev_filter(double *dev_matrix, double *dev_kopia, double *dev_mask, int width, int height, int channels, int maskLength, double *dev_sample, double normalize){
	int x = blockIdx.x*blockDim.x + threadIdx.x;		// przypisanie wspolprzednej do aktualnego watku
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int offset = x + y * gridDim.x;						// numer watka w 1D

	int sampleX = 0;
	int sampleY = 0;
	double result = 0;
	int divMaskLength = (maskLength / 2)-1;				// rozmiar maski liczony od srodka, czyli obrabianego piksela. Potrzeba do p�tli ktora mnozy maske oraz probke.

	if ((x != 0) && (y != 0) && (x != width - 1) && (y != height - 1)) {	// Petla glowna, nie brane sa pod uwage piksele po zewnetrznej stronie obrazka 

		for (int xx = -divMaskLength; xx < (maskLength - divMaskLength); xx++) {		// tworzona jest probka o wielkosci rownej masce, nastepnie kazdy element jest mnozony 
			for (int yy = -divMaskLength; yy < (maskLength - divMaskLength); yy++) {
				dev_sample[sampleY + sampleX * maskLength] = dev_kopia[offset + yy + xx * width];
				result += dev_sample[sampleY + sampleX * maskLength] * dev_mask[sampleY + sampleX * maskLength];
				sampleY++;
			}
			sampleX++;
			sampleY = 0;
		}
		if (normalize != 0) result /= normalize;		// normalizacja

		if (result >= 1) result = 1;
		if (result <= 0) result = 0;
		dev_matrix[offset] = result;
		__syncthreads();
	}
}


int main(void) {

	int filterType = init();
	int maskLength = getMaskLength(filterType);

	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	int originalColor = 7;
	int CPUcolor = 9;
	int GPUcolor = 2;

	time_t start;
	time_t stop;
	double timeResult = 0;

	CImg<double> image("biedronka.jpg");
	CImgDisplay display(image, "Oryginal");

	int width = image.width();
	int height = image.height();
	int channels = image.spectrum();

	CImg<double> output(width, height, 1, channels);
	CImg<double> outputCPU(width, height, 1, channels);

	double *matrix = new double[width * height * channels];
	double *dev_matrix;
	double *kopia = new double[width * height * channels];
	double *dev_kopia;		

	double *mask = new double[maskLength * maskLength];
	double *dev_mask;
	double *sample = new double[maskLength * maskLength];
	double *dev_sample;


	writeOrGetMatrix(image, matrix, WRITE_FLATTEN_MATRIX);
	writeOrGetMatrix(image, kopia, WRITE_FLATTEN_MATRIX);

	setFilter(mask, filterType);

	double normalize = getNormalize(mask, maskLength);


	// ********************* CPU *************************
	// ===================================================

	start = clock();

	filter(matrix, kopia, mask, width, height, channels, maskLength, normalize);

	stop = clock();
	timeResult = (double)(stop - start) / CLOCKS_PER_SEC;

	SetConsoleTextAttribute(hConsole, CPUcolor);
	cout << " CZAS CPU: ";
	SetConsoleTextAttribute(hConsole, originalColor);
	cout << timeResult << "s";
	cout << endl << endl;

	writeOrGetMatrix(outputCPU, matrix, GET_FLATTEN_MATRIX);

	CImgDisplay display2(outputCPU, "CPU");

	writeOrGetMatrix(image, matrix, WRITE_FLATTEN_MATRIX);
	writeOrGetMatrix(image, kopia, WRITE_FLATTEN_MATRIX);

	// ********************* GPU *************************
	// ===================================================

	int sizeX = width;
	int sizeY = height * channels;


	int TILE = 1;

	dim3 block(TILE, TILE);

	int grid_x = sizeX;
	int grid_y = sizeY;

	dim3 grid(grid_x, grid_y);

	// ************* ALOKACJA PAMIECI NA GPU *************
	// ===================================================

	HANDLE_ERROR(hipMalloc((void**)&dev_matrix, width * height * channels * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_kopia, width * height * channels * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_mask, maskLength * maskLength * sizeof(double)));
	HANDLE_ERROR(hipMalloc((void**)&dev_sample, maskLength * maskLength * sizeof(double)));

	// ************* KOPIOWANIE PAMIECI DO GPU *************
	// =====================================================

	HANDLE_ERROR(hipMemcpy(dev_matrix, matrix, width * height * channels * sizeof(double), hipMemcpyHostToDevice));	// kopiowanie do GPU
	HANDLE_ERROR(hipMemcpy(dev_kopia, matrix, width * height * channels * sizeof(double), hipMemcpyHostToDevice));	// kopiowanie do GPU
	HANDLE_ERROR(hipMemcpy(dev_mask, mask, maskLength * maskLength * sizeof(double), hipMemcpyHostToDevice));			// kopiowanie do GPU
	HANDLE_ERROR(hipMemcpy(dev_sample, sample, maskLength * maskLength * sizeof(double), hipMemcpyHostToDevice));			// kopiowanie do GPU

	// **************** WYWOLANIE KERNELA ******************
	// =====================================================

	start = clock();

	dev_filter <<<grid, block >>> (dev_matrix, dev_kopia, dev_mask, width, height, channels, maskLength, dev_sample, normalize);

	stop = clock();
	timeResult = (double)(stop - start) / CLOCKS_PER_SEC;

	SetConsoleTextAttribute(hConsole, GPUcolor);
	cout << " CZAS GPU: ";
	SetConsoleTextAttribute(hConsole, originalColor);
	cout << timeResult << "s";
	cout << endl << endl;

	// ************* KOPIOWANIE PAMIECI DO CPU *************
	// =====================================================

	HANDLE_ERROR(hipMemcpy(matrix, dev_matrix, width * height * channels * sizeof(double), hipMemcpyDeviceToHost));	// kopiowanie z GPU do CPU


	writeOrGetMatrix(output, matrix, GET_FLATTEN_MATRIX);

	CImgDisplay display3(output, "GPU");

	while (!(display.is_closed() && display2.is_closed() && display3.is_closed())){
		display.wait();
		display2.wait();
		display3.wait();
	}

	// **************** ZWOLNIENIE PAMIECI *****************
	// =====================================================

	delete[] mask;
	delete[] matrix;
	delete[] kopia;
	delete[] sample;

	hipFree(dev_matrix);
	hipFree(dev_kopia);
	hipFree(dev_mask);

	return 0;
}


void writeOrGetMatrix(CImg<double> &image, double *matrix, bool type) {

	int width = image.width();
	int height = image.height();
	int channels = image.spectrum();

	for (int c = 0; c < channels; c++) {
		for (int y = 0; y < height; y++) {
			for (int x = 0; x < width; x++) {
				if(!type) matrix[x + y * width + c * width * height] = image(x, y, c)/255;
				else image(x, y, c) = matrix[x + y * width + c * width * height];
			}
		}
	}
}


double getNormalize(double *mask, int size) {

	double result = 0;

	for (int i = 0; i < size * size; i++) {
		result += mask[i];
	}
	return result;
}

void blow(double *mask, int size) {
	for (int i = 0; i < size * size; i++) {
		mask[i] = 1;
	}
}

void sharpen(double *mask) {

	mask[0] = 0;
	mask[1] = -1;
	mask[2] = 0;
	mask[3] = -1;
	mask[4] = 5;
	mask[5] = -1;
	mask[6] = 0;
	mask[7] = -1;
	mask[8] = 0;

}

void prominence(double *mask) {

	mask[0] = -1;
	mask[1] = 0;
	mask[2] = 1;
	mask[3] = -1;
	mask[4] = 1;
	mask[5] = 1;
	mask[6] = -1;
	mask[7] = 0;
	mask[8] = 1;
}
void gradient(double *mask) {

	mask[0] = 0;
	mask[1] = 0;
	mask[2] = 0;
	mask[3] = -1;
	mask[4] = 1;
	mask[5] = 0;
	mask[6] = 0;
	mask[7] = 0;
	mask[8] = 0;

}

void laplace(double *mask) {

	mask[0] = -1;
	mask[1] = -1;
	mask[2] = -1;
	mask[3] = -1;
	mask[4] = 8;
	mask[5] = -1;
	mask[6] = -1;
	mask[7] = -1;
	mask[8] = -1;

}

void filter(double *matrix, double *copy, double *mask, int width, int height, int channels, int maskLength, double normalize) {

	double result = 0;
	int sampleX = 0;
	int sampleY = 0;
	double divMaskLength = floor(maskLength / 2);
	int index = 0;
	double *sample = new double[maskLength * maskLength];
	int cnt = 0;
	double mnozenie = 0;

	for (int k = 0; k < 3; k++) {
		for (int i = divMaskLength; i < (height - divMaskLength); i++) {
			for (int j = divMaskLength; j < (width - divMaskLength); j++) {
				index = j + (i * width) + (k * width * height);
				for (int x = -divMaskLength; x < (maskLength - divMaskLength); x++) {
					for (int y = -divMaskLength; y < (maskLength - divMaskLength); y++) {
						sample[sampleY + sampleX * maskLength] = copy[index + y + x * width];
						mnozenie = (sample[sampleY + sampleX * maskLength] * mask[sampleY + sampleX * maskLength]);
						result = result + mnozenie;
						sampleY++;

					}
					sampleX++;
					sampleY = 0;
				}

				if(normalize != 0) result = result / normalize;
				if (result >= 1) result = 1;
				if (result <= 0) result = 0;
				matrix[index] = result;

				result = 0;
				sampleX = 0;
				sampleY = 0;
			}
		}
	}
	delete[] sample;
}


int init() {

	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	int availableColor = 16;
	int originalColor = 7;
	int setColor = 14;
	int errorColor = 12;

	int maskLength;
	string text;
	int number;
	int filtersCount = 5;

	string filtres[5] = {
		" * (1) ROZMYCIE",
		" * (2) WYOSTRZENIE",
		" * (3) UWYDATNIENIE",
		" * (4) GRADIENT",
		" * (5) LAPLACE"
	};

	do {
		system("cls");

		SetConsoleTextAttribute(hConsole, availableColor);
		cout << " Dostepne filtry: " << endl;
		SetConsoleTextAttribute(hConsole, originalColor);
		cout << "---------------------------" << endl;

		cout << filtres[0] << endl;
		cout << filtres[1] << endl;
		cout << filtres[2] << endl;
		cout << filtres[3] << endl;
		cout << filtres[4] << endl;

		cout << "---------------------------";
		cout << endl << endl;

		cout << " Wprowadz numer: ";
		cin >> text;
		number = atoi(text.c_str());
		number -= 1;
		if (!(number > -1 && number < filtersCount)) {
			SetConsoleTextAttribute(hConsole, errorColor);
			cout << " error!";
			SetConsoleTextAttribute(hConsole, originalColor);
			Sleep(1000);
		}
		else {
			Sleep(250);
		}
	} while (!(number > -1 && number < filtersCount));

	cout << "---------------------------" << endl;

	cout << endl << endl;

	system("cls");
	SetConsoleTextAttribute(hConsole, availableColor);
	cout << " Dostepne filtry: " << endl;
	SetConsoleTextAttribute(hConsole, originalColor);
	cout << "---------------------------" << endl;

	for (int i = 0; i < filtersCount; i++) {
		if (i == number) {
			SetConsoleTextAttribute(hConsole, setColor);
			cout << filtres[i] << endl;
		}
		else {
			SetConsoleTextAttribute(hConsole, originalColor);
			cout << filtres[i] << endl;
		}
	}
	SetConsoleTextAttribute(hConsole, originalColor);

	cout << "---------------------------";
	cout << endl << endl;

	return number + 1;
}

int getMaskLength(int filterType) {
	switch (filterType) {
	case 1:
		return 5;
		break;
	default:
		return 3;
		break;
	}
}

void setFilter(double *mask, int filterType) {

	switch (filterType) {
	case 1:
		blow(mask, 5);
		break;
	case 2:
		sharpen(mask);
		break;
	case 3:
		prominence(mask);
		break;
	case 4:
		gradient(mask);
		break;
	case 5:
		laplace(mask);
		break;
	}
}


